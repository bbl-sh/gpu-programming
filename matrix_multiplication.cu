//%%cuda


/*
Step 1: Allocate host memory
Step 2: Initialize host array
Step 3: Allocate devide 
Step 4: Copy host array to device
Step 5: Setup kernel
Step 6: Launch kernel
Step 7: Check kernel launch errors
Step 8: Sync 
Step 9: Copy result to device
Step 10: Verify the results
*/ 


// CPU function for vector addition

#include <hip/hip_runtime.h>
#include <stdio.h>

void vecAddition(float *h_A, float *h_B, float *h_C, int n){
    for (int i = 0; i < n; i++){
        h_C[i] = h_A[i] + h_B[i];
    }
}

// Cuda function for doing the same 
__global__ void vecAdditionCuda(float *c_A, float *c_B, float *c_C, int n){
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if(i < n){
        c_C[i] = c_A[i] + c_B[i];
    }
}

int main(){
    // Size of the vectors
    const int N = 1024; 
    size_t size = N * sizeof(float);

    // Allocate host memory
    float *h_A = (float*)malloc(size);
    float *h_B = (float*)malloc(size);
    float *h_C = (float*)malloc(size);

    // Initialize host arrays
    for (int i = 0; i < N; i++) {
        h_A[i] = (float)i;
        h_B[i] = (float)(2 * i);
    }

    // Allocate device 
    float *d_A, *d_B, *d_C;
    hipError_t err;

    err = hipMalloc((void**)&d_A, size);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMalloc((void**)&d_B, size);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMalloc((void**)&d_C, size);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

   // Copy host arrays to device
    err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy vector B from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    // Set up kernel launch parameters
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    // Launch the kernel
    vecAdditionCuda<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    // Check for kernel launch errors
    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to launch vectorAdditionCuda kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
// Synchronize to ensure kernel execution is complete
    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA error after kernel launch: %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy result from device to host
    err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Verify the result (print first 10 elements)
    for (int i = 0; i < 10; i++) {
        printf("%f + %f = %f\n", h_A[i], h_B[i], h_C[i]);
    }

    // Free memory
    free(h_A);
    free(h_B);
    free(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}